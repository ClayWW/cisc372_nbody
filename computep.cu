#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

vector3* vals;
vector3** accels;

__global__ void paccel(vector3* vals, vector3** accels, vector3* d_vel, vector3* d_pos, double* d_mass){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k;
    if(i < NUMENTITIES && j < NUMENTITIES){
        if(i == j){
            FILL_VECTOR(accels[i][j],0,0,0);
        }else{
            vector3 distance;
            for (k=0;k<3;k++) distance[k]=d_pos[i][k]-d_pos[j][k];
			double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
			double magnitude=sqrt(magnitude_sq);
			double accelmag=-1*GRAV_CONSTANT*d_mass[j]/magnitude_sq;
			FILL_VECTOR(accels[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
        }
    }

}

__global__ void psum(vector3 *hVel, vector3* hPos, vector3** accels, vector3* accel_sum){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j,k;
    if(i < NUMENTITIES){
        FILL_VECTOR(accel_sum[i],0,0,0);
		for (j=0;j<NUMENTITIES;j++){
			for (k=0;k<3;k++){
				accel_sum[k]+=accels[(i*NUMENTITIES)+j][k];
            }
		}
		for (k=0;k<3;k++){
			hVel[i][k]+=accel_sum[i][k]*INTERVAL;
			hPos[i][k]=hVel[i][k]*INTERVAL;
		}
	}
}


void compute(){
    vector3 *d_hVel;
    vector3 *d_hPos;
    vector3 *d_acc;
    vector3* d_sum;
    double d_mass;
    int blocks = ceilf(NUMENTITIES/16.0f);  //defining our blocks and threads
    int threads = ceilf(NUMENTITIES/(float)blocks);
    dim3 fullgrid(blocks, blocks, 1);
    dim3 blockdim(threads, threads, 1);
    hipMallocManaged((void**) &d_hPos, sizeof(vector3)*NUMENTITIES); //allocating mem for position, velocity, mass, our acceleration and sum functions
    hipMallocManaged((void**) &d_hVel, sizeof(vector3)*NUMENTITIES);
    hipMallocManaged((void**) &d_mass, sizeof(double)*NUMENTITIES);
    hipMallocManaged((void**) &d_acc, sizeof(vector3)*NUMENTITIES);        
    hipMallocManaged((void**) &d_sum, sizeof(vector3)*NUMENTITIES);
    hipMemcpy(d_hPos, hPos, sizeof(vector3)*NUMENTITIES, hipMemcpyHostToDevice); //copying data from host to device memory
    hipMemcpy(d_hVel, hVel, sizeof(vector3)*NUMENTITIES, hipMemcpyHostToDevice);
    hipMemcpy(d_mass, mass, sizeof(double)*NUMENTITIES, hipMemcpyHostToDevice);
    paccel<<<fullgrid, blockdim>>>(d_hPos,d_acc,d_mass); //compute accelerations in parallel
    cudaDeviceSynchronization();
    psum<<<fullgrid.x, blockdim.x>>>(d_acc, d_sum, d_hPos, d_hVel); //sum in parallel
    hipMemcpy(hPos, d_hPos, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost); //copy from device to host memory
    hipMemcpy(hVel, d_hVel, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);
    hipFree(d_hPos); //free everything that was allocated
    hipFree(d_hVel);
    hipFree(d_mass);
    hipFree(d_acc);
    hipFree(d_sum);
}
